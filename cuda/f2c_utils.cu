#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include "f2c.h"
#include <math.h>

__device__ double pow_dd(doublereal *ap, doublereal *bp)
{
	return pow(*ap, *bp);
}

__device__ double pow_ri(real *ap, integer *bp)
{
	double pow, x;
	integer n;
	unsigned long u;

	pow = 1;
	x = *ap;
	n = *bp;

	if (n != 0) {
		if (n < 0) {
			n = -n;
			x = 1 / x;
		}

		for (u = n; ; ) {
			if (u & 01)
				pow *= x;
			if (u >>= 1)
				x *= x;
			else
				break;
		}
	}

	return(pow);
}

__device__ double r_sign(real *a, real *b)
{
	double x;
	x = (*a >= 0 ? *a : -*a);
	return (*b >= 0 ? x : -x);
}

#ifdef __cplusplus
}
#endif

