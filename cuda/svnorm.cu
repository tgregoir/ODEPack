#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include "f2c.h"

/* DECK SVNORM */
__device__ doublereal svnorm_(integer *n, real *v, real *w)
{
    /* System generated locals */
    integer i__1;
    real ret_val, r__1;

    /* Builtin functions */
    //double sqrt(doublereal);

    /* Local variables */
    integer i__;
    real sum;

/* ***BEGIN PROLOGUE  SVNORM */
/* ***SUBSIDIARY */
/* ***PURPOSE  Weighted root-mean-square vector norm. */
/* ***TYPE      SINGLE PRECISION (SVNORM-S, DVNORM-D) */
/* ***AUTHOR  Hindmarsh, Alan C., (LLNL) */
/* ***DESCRIPTION */

/*  This function routine computes the weighted root-mean-square norm */
/*  of the vector of length N contained in the array V, with weights */
/*  contained in the array W of length N: */
/*    SVNORM = SQRT( (1/N) * SUM( V(i)*W(i) )**2 ) */

/* ***SEE ALSO  SLSODE */
/* ***ROUTINES CALLED  (NONE) */
/* ***REVISION HISTORY  (YYMMDD) */
/*   791129  DATE WRITTEN */
/*   890501  Modified prologue to SLATEC/LDOC format.  (FNF) */
/*   890503  Minor cosmetic changes.  (FNF) */
/*   930809  Renamed to allow single/double precision versions. (ACH) */
/* ***END PROLOGUE  SVNORM */
/* **End */

/* ***FIRST EXECUTABLE STATEMENT  SVNORM */
    /* Parameter adjustments */
    --w;
    --v;

    /* Function Body */
    sum = 0.f;
    i__1 = *n;
    for (i__ = 1; i__ <= i__1; ++i__) {
/* L10: */
/* Computing 2nd power */
	r__1 = v[i__] * w[i__];
	sum += r__1 * r__1;
    }
    ret_val = sqrt(sum / *n);
    return ret_val;
/* ----------------------- END OF FUNCTION SVNORM ------------------------ */
} /* svnorm_ */

#ifdef __cplusplus
}
#endif

